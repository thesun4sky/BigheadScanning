#include "hip/hip_runtime.h"
#include <windows.h>
#include <stdio.h>
#include <math.h>
#include <GL/gl.h>
#include <GL/glut.h>
#include <GL/glu.h>
//#include "Vector3D.h"

#include "hip/hip_runtime.h"
#include ""
    
void CreateTable(float* arr,int min, int max, int rgb);
void ScanWithCuda();
void CudaUpdate();
__global__ void Scanning_Kernal(unsigned char* vol, unsigned char* val);
__global__ void Scanning_Kernal2(unsigned char* vol, unsigned char* val, float* dev_Dir, float* dev_Up, float* dev_Cross, float* dev_Eye, int Width, int Height, int Depth);


GLubyte map[256*256*4];
float Eye[3] = {0, 0, 0};
bool EyeHasChanged = false;
int Min, Max;

const int Width = 256;
const int Height = 256;
const int Depth = 225;

bool Interpolation=false;

__constant__ int cWidth;
__constant__ int cHeight;
__constant__ int cDepth;
__constant__ bool cInterpolation;

__constant__ float cEye[3];
__constant__ float cU[3];
__constant__ float cCross[3];
__constant__ float cDir[3];
__constant__ float cColor_r[256];
__constant__ float cColor_g[256];
__constant__ float cColor_b[256];
__constant__ float cAlpha[256];


    const int MapSize = Width*Height*Depth*sizeof(unsigned char);
	hipEvent_t start, stop;//Ÿ���̺�Ʈ
	float Time;
    unsigned char *vol = new unsigned char[Width * Height * Depth];
    unsigned char *val = new unsigned char[Width * Height];
   
    float Look[3] = {128,128,112.5};
	float Up[3] = {0, 1, 0};
	float Cross[3] = {0, 0, 0};
	float Dir[3] = {0, 0, 0};
	float u[3] = {0, 0, 0};
    
    float Color_r[256];
    float Color_g[256];
    float Color_b[256];
    float Alpha[256];
	float *dev_Eye;
	float *dev_Dir;
	float *dev_U;
	float *dev_Cross;
	unsigned char* dev_vol;
	unsigned char* dev_val;


void myreshape(int w, int h)
{
 
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glEnable(GL_BLEND);
	glEnable(GL_TEXTURE_2D);
 
	GLuint texId;
	glGenTextures(1, &texId);
	glBindTexture(GL_TEXTURE_2D, texId);
 
	glTexImage2D(
		GL_TEXTURE_2D, 0, GL_RGBA,
		Width, Height, 0,
		GL_RGBA, GL_UNSIGNED_BYTE,
		map
		);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexEnvi(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);
 
 
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glOrtho(0, Width, 0, Height, 0, 1);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	
	glBindTexture(GL_TEXTURE_2D, texId);
}

void mykeyboard(unsigned char keyPressed, int x, int y)
{ 
 switch (keyPressed)//Ű�� ����� ��ġ �̵�
 {
 case 'a'://left
  Eye[0]-=5;
  EyeHasChanged = true;
  break;
 case 'd'://right
  Eye[0]+=5;
  EyeHasChanged = true;
  break;
 case 'w'://up
  Eye[1]+=5;
  EyeHasChanged = true;
  break;
 case 's'://down
  Eye[1]-=5;
  EyeHasChanged = true;
  break;
 case 'q'://down
  Eye[2]-=5;
  EyeHasChanged = true;
  break;
 case 'e'://down
  Eye[2]+=5;
  EyeHasChanged = true;
  break;
 case 'z'://down
  Min-=5;
  Max-=5;
  break;
 case 'x'://down
  Min+=5;
  Max+=5;
  break;
 case 'A'://left
  Eye[0]-=10;
  EyeHasChanged = true;
  break;
 case 'D'://right
  Eye[0]+=10;
  EyeHasChanged = true;
  break;
 case 'W'://up
  Eye[1]+=10;
  EyeHasChanged = true;
  break;
 case 'S'://down
  Eye[1]-=10;
  EyeHasChanged = true;
  break;
 case 'Q'://down
  Eye[2]-=10;
  EyeHasChanged = true;
  break;
 case 'E'://down
  Eye[2]+=10;
  EyeHasChanged = true;
  break;
 case 'Z'://down
  Min-=10;
  Max-=10;
  break;
 case 'X'://down
  Min+=10;
  Max+=10;
  break;
case '['://ZoomIn
  Cross[0]*=0.8;
  Cross[1]*=0.8;
  Cross[2]*=0.8;
  u[0]*=0.8;
  u[1]*=0.8;
  u[2]*=0.8;
  EyeHasChanged = false;
  break;
 case ']'://ZoomOut
  Cross[0]*=1.2;
  Cross[1]*=1.2;
  Cross[2]*=1.2;
  u[0]*=1.2;
  u[1]*=1.2;
  u[2]*=1.2;
  EyeHasChanged = false;
  break;
 }
 CudaUpdate();
 glutPostRedisplay();
}

void MyMouseWheelFunc(int wheel, int direction, int x, int y)
{
    if(direction > 0)
    {
        Min-=10;
        Max-=10;
    }
    else   
    {
        Min+=10;
        Max+=10;
    }

 CudaUpdate();
 glutPostRedisplay();

}

void MyDisplay() {

    
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glEnable(GL_BLEND);
	glEnable(GL_TEXTURE_2D);
 
	GLuint texId;
	glGenTextures(1, &texId);
	glBindTexture(GL_TEXTURE_2D, texId);
 
	glTexImage2D(
		GL_TEXTURE_2D, 0, GL_RGBA,
		Width, Height, 0,
		GL_RGBA, GL_UNSIGNED_BYTE,
		map
		);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexEnvi(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);
 
 
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glOrtho(0, Width, 0, Height, 0, 1);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	
	glBindTexture(GL_TEXTURE_2D, texId);
 
	glBegin(GL_QUADS);
	glTexCoord2f(0.0, 0.0); glVertex2f(0, Height);
	glTexCoord2f(1.0, 0.0); glVertex2f(Width, Height);
	glTexCoord2f(1.0, 1.0); glVertex2f(Width, 0);
	glTexCoord2f(0.0, 1.0); glVertex2f(0, 0);
	glEnd();
	
	glFlush();
}

int main(int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(600, 600);
	glutInitWindowPosition(200, 200);
	glutCreateWindow("openGL Sample Drawing");
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
    ScanWithCuda();
	glutDisplayFunc(MyDisplay);
    //glutReshapeFunc(myreshape);
    glutKeyboardFunc(mykeyboard);
    //glutMouseWheelFunc(MyMouseWheelFunc);
	glutMainLoop();
	return 0;
}

void ScanWithCuda(){
    
    
    int Choice;
    
    hipError_t cudaStatus;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    
    
	//����� �ҷ�����
	FILE *fp = fopen("C:/Bighead.den", "rb");
	fread(vol, Width*Height*Depth, 1, fp);
	fclose(fp);
	

    printf("Input eye position and min, max\nx y z min max\n");
	scanf("%f %f %f %d %d", &Eye[0], &Eye[1], &Eye[2], &Min, &Max);
	
    CreateTable(Color_r,Min,Max,1);
    CreateTable(Color_g,Min,Max,2);
    CreateTable(Color_b,Min,Max,3);
    CreateTable(Alpha,Min,Max,0);
    
    printf("What you want to try?\n1. __constant__\n2. __global__\n");
	scanf("%d", &Choice);
	
    int inter;
    printf("Do you want to Interpolate?\n1. yes \n2. no\n");
    scanf("%d",&inter);
    if(inter == 1) Interpolation = true;
	Dir[0] = Look[0] - Eye[0];
	Dir[1] = Look[1] - Eye[1];
	Dir[2] = Look[2] - Eye[2];
    //Normalization
	float scalar = sqrt( (Dir[0] * Dir[0]) + (Dir[1] * Dir[1]) + (Dir[2] * Dir[2]) );
	for(int i=0; i<3; i++){
		Dir[i] /= scalar;
	}
	//Cross
	Cross[0] = Up[1]*Dir[2] - Up[2]*Dir[1];
	Cross[1] = Up[2]*Dir[0] - Up[0]*Dir[2];
	Cross[2] = Up[0]*Dir[1] - Up[1]*Dir[0];
    //Normalization
    scalar = sqrt( (Cross[0] * Cross[0]) + (Cross[1] * Cross[1]) + (Cross[2] * Cross[2]) );
	for(int i=0; i<3; i++){
		Cross[i] /= scalar;
	}
	//U
	u[0] = Dir[1]*Cross[2] - Dir[2]*Cross[1];
	u[1] = Dir[2]*Cross[0] - Dir[0]*Cross[2];
	u[2] = Dir[0]*Cross[1] - Dir[1]*Cross[0];

	hipMalloc((void**)&dev_vol, MapSize);
	hipMalloc((void**)&dev_val, Width*Height*4*sizeof(unsigned char));
	hipMalloc((void**)&dev_Eye, 3*sizeof(float));
	hipMalloc((void**)&dev_Dir, 3*sizeof(float));
	hipMalloc((void**)&dev_U, 3*sizeof(float));
	hipMalloc((void**)&dev_Cross, 3*sizeof(float));


	hipMemcpy(dev_vol, vol, MapSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_Eye, Eye, 3*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Dir, Dir, 3*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_U, u, 3*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Cross, Cross, 3*sizeof(float), hipMemcpyHostToDevice);

    //��� ������ ����
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cWidth),&Width, sizeof(int));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cHeight),&Height, sizeof(int));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cDepth),&Depth, sizeof(int));

    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cInterpolation),&Interpolation, sizeof(bool));

    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cEye),Eye, 3*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cU),u, 3*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCross),Cross, 3*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cDir),Dir, 3*sizeof(float));

    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cColor_r),Color_r, 256*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cColor_g),Color_g, 256*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cColor_b),Color_b, 256*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cAlpha),Alpha, 256*sizeof(float));
    if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpyToSymbol failed!\n");
            //goto Error;
        }

	hipEventRecord(start,0);//�ð����� ��ŸƮ
    (Choice == 1) ? Scanning_Kernal<<<256,256>>>(dev_vol, dev_val) : Scanning_Kernal2<<<256,256>>>(dev_vol, dev_val, dev_Dir, dev_U, dev_Cross, dev_Eye, Width, Height, Depth);
	hipEventRecord(stop,0);//�ð����� ����
	hipEventSynchronize(stop);
	hipEventElapsedTime(&Time,start,stop);

	hipMemcpy(map, dev_val, Width*Height*4*sizeof(unsigned char), hipMemcpyDeviceToHost);
    
	

	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("%d�� Ŀ���Լ� �ҿ�ð� : %f msec\n",Choice, Time);

}


void CreateTable(float* arr,int min, int max, int rgb){
    int i=0, bone_min=50, bone_max=100;
    float j=1/(float)(max-min);
    if(rgb == 0){
        for( ;i<min; i++)
             arr[i] = 0;
        for( ;i<max; i++)
             arr[i] = arr[i-1]+j;
         for( ;i<256; i++)
             arr[i] = 1;
    }
    if(rgb == 1){
        for( ;i<bone_min; i++)
             arr[i] = 0;
        for( ;i<bone_max; i++)
             arr[i] = 0.9;
         for( ;i<256; i++)
             arr[i] = 1.0;
    }
    if(rgb == 2){
        for( ;i<bone_min; i++)
             arr[i] = 0;
        for( ;i<bone_max; i++)
             arr[i] = 0.3;
         for( ;i<256; i++)
             arr[i] = 1.0;
    }
    if(rgb == 3){
        for( ;i<bone_min; i++)
             arr[i] = 0;
        for( ;i<bone_max; i++)
             arr[i] = 0.1;
         for( ;i<256; i++)
             arr[i] = 0.2;
    }
}

void CudaUpdate()
{

    hipError_t cudaStatus;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    
    CreateTable(Color_r,Min,Max,1);
    CreateTable(Color_g,Min,Max,2);
    CreateTable(Color_b,Min,Max,3);
    CreateTable(Alpha,Min,Max,0);

    //����ġ ���濡 ���� ������Ʈ
    if(EyeHasChanged){
        Dir[0] = Look[0] - Eye[0];
	    Dir[1] = Look[1] - Eye[1];
	    Dir[2] = Look[2] - Eye[2];

        //Normalization
	    float scalar = sqrt( (Dir[0] * Dir[0]) + (Dir[1] * Dir[1]) + (Dir[2] * Dir[2]) );
	    for(int i=0; i<3; i++){
		    Dir[i] /= scalar;
	    }
	    //Cross
	    Cross[0] = Up[1]*Dir[2] - Up[2]*Dir[1];
	    Cross[1] = Up[2]*Dir[0] - Up[0]*Dir[2];
	    Cross[2] = Up[0]*Dir[1] - Up[1]*Dir[0];
        //Normalization
        scalar = sqrt( (Cross[0] * Cross[0]) + (Cross[1] * Cross[1]) + (Cross[2] * Cross[2]) );
	    for(int i=0; i<3; i++){
		    Cross[i] /= scalar;
	    }
	    //U
	    u[0] = Dir[1]*Cross[2] - Dir[2]*Cross[1];
	    u[1] = Dir[2]*Cross[0] - Dir[0]*Cross[2];
	    u[2] = Dir[0]*Cross[1] - Dir[1]*Cross[0];

        EyeHasChanged == false;
    }

	hipMemcpy(dev_Eye, Eye, 3*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Dir, Dir, 3*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_U, u, 3*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Cross, Cross, 3*sizeof(float), hipMemcpyHostToDevice);

    //��� ������ ����
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cEye),Eye, 3*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cU),u, 3*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cCross),Cross, 3*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cDir),Dir, 3*sizeof(float));

    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cColor_r),Color_r, 256*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cColor_g),Color_g, 256*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cColor_b),Color_b, 256*sizeof(float));
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cAlpha),Alpha, 256*sizeof(float));
    if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpyToSymbol failed!\n");
            //goto Error;
    }

	hipEventRecord(start,0);//�ð����� ��ŸƮ
    Scanning_Kernal<<<256,256>>>(dev_vol, dev_val);
	hipEventRecord(stop,0);//�ð����� ����
	hipEventSynchronize(stop);
	hipEventElapsedTime(&Time,start,stop);

	hipMemcpy(map, dev_val, Width*Height*4*sizeof(unsigned char), hipMemcpyDeviceToHost);
    
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("������Ʈ Ŀ���Լ� �ҿ�ð� : %f msec\n", Time);

}

__device__ unsigned char Interpolation_Kernal(float* posf, unsigned char* vol) {
    
    if(posf[2] > 220) return 0;
    float back_a = vol[(cWidth*cHeight)*(int)posf[2] + cWidth*((int)posf[1]+1) + (int)posf[0]];
    float back_b = vol[(cWidth*cHeight)*(int)posf[2] + cWidth*((int)posf[1]+1) + (int)posf[0]+1];
    float back = back_a * (1.0 - (posf[0] - (int)posf[0])) + back_b * (posf[0] - (int)posf[0]);  //�� ����

    float front_a = vol[(cWidth*cHeight)*((int)posf[2]+1) + cWidth*((int)posf[1]+1) + (int)posf[0]];
    float front_b = vol[(cWidth*cHeight)*((int)posf[2]+1) + cWidth*((int)posf[1]+1) + (int)posf[0]+1];
    float front = front_a * (1.0 - (posf[0] - (int)posf[0])) + front_b * (posf[0] - (int)posf[0]);  // �� ����

    float UpSide = back * (1.0 - (posf[1] - (int)posf[1])) + front * (posf[1] - (int)posf[1]);  //��
    

    back_a = vol[(cWidth*cHeight)*(int)posf[2] + cWidth*((int)posf[1]) + (int)posf[0]];
    back_b = vol[(cWidth*cHeight)*(int)posf[2] + cWidth*((int)posf[1]) + (int)posf[0]+1];
    back = back_a * (1.0 - (posf[0] - (int)posf[0])) + back_b * (posf[0] - (int)posf[0]); //�Ʒ� ����

    front_a = vol[(cWidth*cHeight)*((int)posf[2]+1) + cWidth*((int)posf[1]) + (int)posf[0]];
    front_b = vol[(cWidth*cHeight)*((int)posf[2]+1) + cWidth*((int)posf[1]) + (int)posf[0]+1];
    front = front_a * (1.0 - (posf[0] - (int)posf[0])) + front_b * (posf[0] - (int)posf[0]); //�Ʒ� ����

    float BottomSide = back * (1.0 - (posf[1] - (int)posf[1])) + front * (posf[1] - (int)posf[1]); //�Ʒ�
    
    unsigned char output = UpSide * (1.0 - (posf[2] - (int)posf[2])) + BottomSide * (posf[2] - (int)posf[2]);//���
    return output;
}


__global__ void Scanning_Kernal(unsigned char* vol, unsigned char* val) {

    float a_new=0,a_old=0, r_new=0,r_old=0, g_new=0,g_old=0, b_new=0,b_old=0,posf[3];
	int pos[3],tpos[3];
    float start[3];
	tpos[0] = threadIdx.x-128;
	tpos[1] = blockIdx.x-128;
	tpos[2] = cWidth*(tpos[1]+128)+tpos[0]+128;
	unsigned char d,a_sum=0,c_sum=0;
    start[0] = cEye[0] + cCross[0]*tpos[0] + cU[0]*tpos[1];
    start[1] = cEye[1] + cCross[1]*tpos[0] + cU[1]*tpos[1];
    start[2] = cEye[2] + cCross[2]*tpos[0] + cU[2]*tpos[1];

	for(int k=0;k<500; k++){
		    posf[0] = start[0] + cDir[0]*k;
		    posf[1] = start[1] + cDir[1]*k;
		    posf[2] = start[2] + cDir[2]*k;
		if(posf[0]>=0 && posf[0]<256 && posf[1]>=0 && posf[1]<256 && posf[2]>=0 && posf[2]<225){
            if(cInterpolation) d = Interpolation_Kernal(posf, vol);
			else d = vol[(cWidth*cHeight)*(int)posf[2] + cWidth*(int)posf[1] + (int)posf[0]];
            a_new = a_old + (1-a_old) * cAlpha[d];
            r_new = r_old + (1-a_old) * cColor_r[d] * cAlpha[d];
            g_new = g_old + (1-a_old) * cColor_g[d] * cAlpha[d];
            b_new = b_old + (1-a_old) * cColor_b[d] * cAlpha[d];
            r_old = r_new;
            g_old = g_new;
            b_old = b_new;
            a_old = a_new;
        }
	}
	val[tpos[2]*4+0] = r_new*255;
	val[tpos[2]*4+1] = g_new*255;
	val[tpos[2]*4+2] = b_new*255;
	val[tpos[2]*4+3] = 0xff;
}


__global__ void Scanning_Kernal2(unsigned char* vol, unsigned char* val, float* dev_Dir, float* dev_U, float* dev_Cross, float* dev_Eye, int Width, int Height, int Depth) {

	int pos[3],tpos[3];
	tpos[0] = threadIdx.x-128;
	tpos[1] = blockIdx.x-128;
	tpos[2] = Width*(tpos[1]+128)+tpos[0]+128;
	unsigned char Found_max = 0;

	for(int k=0;k<500; k++){
		pos[0] = dev_Eye[0] + dev_Cross[0]*tpos[0] + dev_U[0]*tpos[1] + dev_Dir[0]*k;
		pos[1] = dev_Eye[1] + dev_Cross[1]*tpos[0] + dev_U[1]*tpos[1] + dev_Dir[1]*k;
		pos[2] = dev_Eye[2] + dev_Cross[2]*tpos[0] + dev_U[2]*tpos[1] + dev_Dir[2]*k;

		if(pos[0]>=0 && pos[0]<256 && pos[1]>=0 && pos[1]<256 && pos[2]>=0 && pos[2]<225)
			(Found_max > vol[(Width*Height)*pos[2] + Width*pos[1] + pos[0]]) ? 1 : Found_max = vol[(Width*Height)*pos[2] + Width*pos[1] + pos[0]];
	}
	val[tpos[2]] = Found_max;
}
